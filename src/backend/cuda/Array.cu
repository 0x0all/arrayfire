#include <cassert>
#include <af/dim4.hpp>
#include <Array.hpp>
#include <iostream>
#include <kernel/elwise.hpp> //set

using af::dim4;

namespace cuda
{
    using std::ostream;

    template<typename T>
    T* cudaMallocWrapper(const size_t &elements)
    {
        T* ptr = NULL;
        //FIXME: Add checks
        hipMalloc(reinterpret_cast<void**>(&ptr), sizeof(T) * elements);
        return ptr;
    }

    // FIXME: Add checks
    template<typename T>
    Array<T>::Array(af::dim4 dims) :
        ArrayInfo(dims, af::dim4(0,0,0,0), calcStrides(dims), (af_dtype)dtype_traits<T>::af_type),
        data(cudaMallocWrapper<T>(dims.elements())),
        parent()
    {}

    // FIXME: Add checks
    template<typename T>
    Array<T>::Array(af::dim4 dims, T val) :
        ArrayInfo(dims, af::dim4(0,0,0,0), calcStrides(dims), (af_dtype)dtype_traits<T>::af_type),
        data(cudaMallocWrapper<T>(dims.elements())),
        parent()
    {
        kernel::set(data, val, elements());
    }

    // FIXME: Add checks
    template<typename T>
    Array<T>::Array(af::dim4 dims, const T * const in_data) :
    ArrayInfo(dims, af::dim4(0,0,0,0), calcStrides(dims), (af_dtype)dtype_traits<T>::af_type),
        data(cudaMallocWrapper<T>(dims.elements())),
        parent()
    {
        hipMemcpy(data, in_data, dims.elements() * sizeof(T), hipMemcpyHostToDevice);
    }

    // FIXME: Add checks
    template<typename T>
    Array<T>::~Array() { hipFree(data); }

    template<typename T>
    const Array<T> &
    getArray(const af_array &arr)
    {
        Array<T> *out = reinterpret_cast<Array<T>*>(arr);
        return *out;
    }

    template<typename T>
    Array<T> &
    getWritableArray(const af_array &arr)
    {
        const Array<T> &out = getArray<T>(arr);
        return const_cast<Array<T>&>(out);
    }

    template<typename T>
    af_array
    getHandle(const Array<T> &arr)
    {
        af_array out = reinterpret_cast<af_array>(&arr);
        return out;
    }

    template<typename T>
    Array<T> *
    createDataArray(const dim4 &size, const T * const data)
    {
        Array<T> *out = new Array<T>(size, data);
        return out;
    }

    template<typename T>
    Array<T>*
    createValueArray(const dim4 &size, const T& value)
    {
        Array<T> *out = new Array<T>(size, value);
        return out;
    }

    template<typename T>
    Array<T>*
    createEmptyArray(const dim4 &size)
    {
        Array<T> *out = new Array<T>(size);
        return out;
    }

    template<typename T>
    Array<T> *
    createSubArray(const Array<T>& parent, const dim4 &dims, const dim4 &offset, const dim4 &stride)
    {
        assert("NOT IMPLEMENTED" && 1 != 1);
        return NULL;
    }

    template<typename T>
    void
    destroyArray(const af_array &arr)
    {
        Array<T> &obj = getWritableArray<T>(arr);
        delete &obj;
    }

#define INSTANTIATE(T)                                                  \
    template       Array<T>&  getWritableArray<T> (const af_array &arr); \
    template const Array<T>&  getArray<T>         (const af_array &arr); \
    template       af_array   getHandle<T>        (const Array<T> &A);  \
    template       Array<T>*  createDataArray<T>  (const dim4 &size, const T * const data); \
    template       Array<T>*  createValueArray<T> (const dim4 &size, const T &value); \
    template       Array<T>*  createEmptyArray<T> (const dim4 &size);   \
    template       Array<T>*  createSubArray<T>       (const Array<T> &parent, const dim4 &dims, const dim4 &offset, const dim4 &stride); \
    template       void       destroyArray<T>     (const af_array &arr); \
    template                  Array<T>::~Array();

    INSTANTIATE(float)
    INSTANTIATE(double)
    INSTANTIATE(cfloat)
    INSTANTIATE(cdouble)
    INSTANTIATE(int)
    INSTANTIATE(uint)
    INSTANTIATE(uchar)
    INSTANTIATE(char)
}
