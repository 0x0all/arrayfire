#include "hip/hip_runtime.h"
#include "reduce_impl.hpp"

namespace cuda
{
    //max
    INSTANTIATE(af_max_t, float  , float  )
    INSTANTIATE(af_max_t, double , double )
    INSTANTIATE(af_max_t, cfloat , cfloat )
    INSTANTIATE(af_max_t, cdouble, cdouble)
    INSTANTIATE(af_max_t, int    , int    )
    INSTANTIATE(af_max_t, uint   , uint   )
    INSTANTIATE(af_max_t, char   , char   )
    INSTANTIATE(af_max_t, uchar  , uchar  )
}
