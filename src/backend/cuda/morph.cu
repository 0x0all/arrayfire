#include <af/dim4.hpp>
#include <af/defines.h>
#include <ArrayInfo.hpp>
#include <Array.hpp>
#include <morph.hpp>
#include <kernel/morph.hpp>
#include <stdexcept>

using af::dim4;

namespace cuda
{

template<typename T, bool isDilation>
Array<T> * morph(const Array<T> &in, const Array<T> &mask)
{
    const dim4 mdims = mask.dims();

    if (mdims[0] != mdims[1])
        throw std::runtime_error("Only square masks are supported in cuda morph currently");
    if (mdims[0] > 19)
        throw std::runtime_error("Upto 19x19 square kernels are only supported in cuda currently");

    Array<T>* out = createEmptyArray<T>(in.dims());

    hipMemcpyToSymbol(HIP_SYMBOL(kernel::cFilter), mask.get(),
                       mdims[0] * mdims[1] * sizeof(T),
                       0, hipMemcpyDeviceToDevice);

    if (isDilation)
        kernel::morph<T, true >(*out, in, mdims[0]);
    else
        kernel::morph<T, false>(*out, in, mdims[0]);

    return out;
}

template<typename T, bool isDilation>
Array<T> * morph3d(const Array<T> &in, const Array<T> &mask)
{
    const dim4 mdims = mask.dims();

    if (mdims[0] != mdims[1] || mdims[0] != mdims[2])
        throw std::runtime_error("Only cube masks are supported in cuda morph currently");
    if (mdims[0] > 7)
        throw std::runtime_error("Upto 7x7x7 kernels are only supported in cuda currently");

    if (in.dims()[3] > 1)
        throw std::runtime_error("Batch not supported for volumetic morphological operations");

    Array<T>* out       = createEmptyArray<T>(in.dims());

    hipMemcpyToSymbol(HIP_SYMBOL(kernel::cFilter), mask.get(),
                       mdims[0] * mdims[1] *mdims[2] * sizeof(T),
                       0, hipMemcpyDeviceToDevice);

    if (isDilation)
        kernel::morph3d<T, true >(*out, in, mdims[0]);
    else
        kernel::morph3d<T, false>(*out, in, mdims[0]);

    return out;
}

#define INSTANTIATE(T)\
    template Array<T> * morph  <T, true >(const Array<T> &in, const Array<T> &mask);\
    template Array<T> * morph  <T, false>(const Array<T> &in, const Array<T> &mask);\
    template Array<T> * morph3d<T, true >(const Array<T> &in, const Array<T> &mask);\
    template Array<T> * morph3d<T, false>(const Array<T> &in, const Array<T> &mask);

INSTANTIATE(float )
INSTANTIATE(double)
INSTANTIATE(char  )
INSTANTIATE(int   )
INSTANTIATE(uint  )
INSTANTIATE(uchar )

}
