#include "hip/hip_runtime.h"
#include <functional>
#include <af/defines.h>
#include <kernel/elwise.hpp>

namespace cuda
{
namespace kernel
{

template<typename T>
__global__
void setKernel(T* ptr, T val, const size_t elements)
{
    const size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < elements) {
        ptr[idx] = val;
    }
}

template<typename T>
void set(T* ptr, T val, const size_t &elements)
{
    dim3 threads(512);
    dim3 blocks(divup(elements,threads.x));
    typedef typename af::dtype_traits<T>::cuda_type     cuType;
    typedef typename af::dtype_traits<T>::cuda_type_ptr cuTypePtr;

    setKernel<cuType><<<blocks, threads>>>(reinterpret_cast<cuTypePtr>(ptr), *reinterpret_cast<cuTypePtr>(&val), elements);
}

template void set<float>(float* ptr, float val, const size_t &elements);
template void set<double>(double* ptr, double val, const size_t &elements);
template void set<cfloat>(cfloat* ptr, cfloat val, const size_t &elements);
template void set<cdouble>(cdouble* ptr, cdouble val, const size_t &elements);
template void set<char>(char* ptr, char val, const size_t &elements);
template void set<int>(int* ptr, int val, const size_t &elements);
template void set<unsigned>(unsigned* ptr, unsigned val, const size_t &elements);
template void set<unsigned char>(unsigned char* ptr, unsigned char val, const size_t &elements);

}
}
