#include <cassert>

#include <hip/hip_runtime_api.h>
#include <af/array.h>
#include <af/defines.h>
#include <Array.hpp>
#include <copy.hpp>

namespace cuda
{

    template<typename T>
    void copyData(T *data, const Array<T> &A)
    {
        //FIXME: Add checks
        hipMemcpy(data, A.get(), A.elements()*sizeof(T), hipMemcpyDeviceToHost);

        return;
    }


    template<typename T>
    Array<T> *copyArray(const Array<T> &A)
    {
        Array<T> *out = createEmptyArray<T>(A.dims());

        // FIXME: Add checks
        hipMemcpy(out->get(), A.get(), A.elements()*sizeof(T), hipMemcpyDeviceToDevice);
        return out;
    }


#define INSTANTIATE(T)                                                  \
    template void      copyData<T> (T *data, const Array<T> &from);     \
    template Array<T>* copyArray<T>(const Array<T> &A);                 \

    INSTANTIATE(float)
    INSTANTIATE(double)
    INSTANTIATE(cfloat)
    INSTANTIATE(cdouble)
    INSTANTIATE(int)
    INSTANTIATE(uint)
    INSTANTIATE(uchar)
    INSTANTIATE(char)
}
